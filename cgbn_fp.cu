#include "hip/hip_runtime.h"

#include "cgbn_fp.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>

#include "cgbn/cgbn.h"
#include "utility/cpu_support.h"
#include "utility/cpu_simple_bn_math.h"
#include "utility/gpu_support.h"

#define TPI 8
typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;
#define max_threads_per_block  (1024/TPI)


namespace gpu{

__device__ void device_fp_add(env_t& bn_env, cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value){
  env_t::cgbn_t tin1, tin2, tmodule_data, tscratch;

  cgbn_load(bn_env, tin1, in1);
  cgbn_load(bn_env, tin2, in2);
  cgbn_load(bn_env, tmodule_data, module_data);

  uint32_t carry = cgbn_add(bn_env, tscratch, tin1, tin2);
  if(carry || cgbn_compare(bn_env, tscratch, tmodule_data) >= 0){
    if(carry){
      env_t::cgbn_t tmax_value;
      cgbn_load(bn_env, tmax_value, max_value);
      cgbn_sub(bn_env, tin1, tmax_value, tmodule_data);
      cgbn_add(bn_env, tin2, tin1, tscratch);
      cgbn_add_ui32(bn_env, tin1, tin2, 1);
    }else{
      cgbn_sub(bn_env, tin1, tscratch, tmodule_data);
    }
    cgbn_store(bn_env, in1, tin1);
  }else{
    cgbn_store(bn_env, in1, tscratch);
  }
}
__global__ void kernel_fp_add(cgbn_error_report_t* report, cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value, const uint32_t count){
  int instance = (blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(instance >= count) return;
  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());
  device_fp_add(bn_env, in1 + instance, in2 + instance, module_data + instance, max_value);
}

__device__ void device_fp_sub(env_t& bn_env, cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value){
  env_t::cgbn_t tin1, tin2, tmodule_data, tscratch;
  cgbn_load(bn_env, tin1, in1);
  cgbn_load(bn_env, tin2, in2);
  cgbn_load(bn_env, tmodule_data, module_data);

  if(cgbn_compare(bn_env, tin1, tin2) < 0){
    uint32_t carry = cgbn_add(bn_env, tscratch, tin1, tmodule_data);
    if(carry > 0){
      env_t::cgbn_t tmax_value;
      cgbn_load(bn_env, tmax_value, max_value);
      cgbn_sub(bn_env, tin1, tmax_value, tin2);
      cgbn_add(bn_env, tmax_value, tin1, tscratch);
      cgbn_add_ui32(bn_env, tin1, tmax_value, 1);
    }else{
      cgbn_sub(bn_env, tin1, tscratch, tin2);
    }
    cgbn_store(bn_env, in1, tin1);
  }else{
      cgbn_sub(bn_env, tscratch, tin1, tin2);
      cgbn_store(bn_env, in1, tscratch);
  }
}

__global__ void kernel_fp_sub(cgbn_error_report_t* report, cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value, const uint32_t count){
  int instance = (blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(instance >= count) return;
  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());
  device_fp_sub(bn_env, in1 + instance, in2 + instance, module_data + instance, max_value);
}

__device__ void device_mul_reduce(const env_t& bn_env, uint32_t* res,cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* tmp_buffer, const uint64_t inv){
  const int group_thread = threadIdx.x & (TPI-1);
  env_t::cgbn_t  tin1, tin2, tmodule_data, tb, tres,tres2, add_res;                                             
  cgbn_load(bn_env, tin1, in1);  
  cgbn_load(bn_env, tin2, in2);   
  cgbn_load(bn_env, tmodule_data, module_data);     

  const int n = BITS/32;
  env_t::cgbn_wide_t tc;
  cgbn_mul_wide(bn_env, tc, tin1, tin2);
  cgbn_store(bn_env, res, tc._low);
  cgbn_store(bn_env, res + n, tc._high);

  for(int i = 0; i < n; i+=2){
    cgbn_load(bn_env, tres, res+i);
    cgbn_load(bn_env, tres2, res+n+i);

    if(group_thread == 0){
      uint64_t *p64 = (uint64_t*)(res+i);
      uint64_t k = inv * p64[0];
      uint32_t *p32 = (uint32_t*)&k;
      tmp_buffer->_limbs[0] = p32[0];
      tmp_buffer->_limbs[1] = p32[1];
      for(int j = 2; j < BITS/32; j++){
        tmp_buffer->_limbs[j] = 0;
      }
    }

    cgbn_load(bn_env, tb, tmp_buffer);      

    env_t::cgbn_wide_t mul_res;
    cgbn_mul_wide(bn_env, mul_res, tmodule_data, tb);

    uint32_t carryout = cgbn_add(bn_env, add_res, mul_res._low, tres);
    cgbn_store(bn_env, res+i, add_res);   
    
    cgbn_store(bn_env, tmp_buffer, mul_res._high);
    if(group_thread == 0){
      uint64_t tmp_carry = ((uint64_t*)tmp_buffer->_limbs)[0];
      tmp_carry += carryout;
      uint32_t *p = (uint32_t*)&tmp_carry;
      tmp_buffer->_limbs[0] = p[0];
      tmp_buffer->_limbs[1] = p[1];
    }

    cgbn_load(bn_env, tb, tmp_buffer);      
    cgbn_add(bn_env, add_res, tres2, tb);
    cgbn_store(bn_env, res+n+i, add_res);   
  }
  cgbn_load(bn_env, tres, res+n);
  if(cgbn_compare(bn_env, tres, tmodule_data) >= 0){
    cgbn_sub(bn_env, tres2, tres, tmodule_data);
    cgbn_store(bn_env, res+n, tres2);
  }
}

__global__ void kernel_mul_reduce(cgbn_error_report_t* report, uint32_t* res,cgbn_mem_t<BITS>* const in1, cgbn_mem_t<BITS>* const in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* tmp_buffer, uint64_t inv, const uint32_t count){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int instance = tid / TPI;
  if(instance >= count) return;

  context_t bn_context(cgbn_report_monitor, report, instance);
  env_t          bn_env(bn_context.env<env_t>());  
  device_mul_reduce(bn_env, res + instance * 3 * BITS/32, in1 + instance, in2 + instance, module_data + instance, tmp_buffer + instance, inv);
}

int fp_add(cgbn_mem_t<BITS>* in1, cgbn_mem_t<BITS>* in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value, const uint32_t count){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  uint32_t instances = std::min(count, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (count + instances - 1) / instances;
  kernel_fp_add<<<blocks, threads>>>(report, in1, in2, module_data, max_value, count);
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
  return 0;
}

int fp_sub(cgbn_mem_t<BITS>* in1, cgbn_mem_t<BITS>* in2, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* max_value, const uint32_t count){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  uint32_t instances = std::min(count, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (count + instances - 1) / instances;
  kernel_fp_sub<<<blocks, threads>>>(report, in1, in2, module_data, max_value, count);
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
  return 0;
}

int fp_mul_reduce(cgbn_mem_t<BITS>* in1, cgbn_mem_t<BITS>* in2, uint64_t inv, cgbn_mem_t<BITS>* module_data, cgbn_mem_t<BITS>* tmp_buffer, uint32_t* res, const uint32_t count){
  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 

  uint32_t instances = std::min(count, (uint32_t)max_threads_per_block);
  uint32_t threads = instances * TPI;
  uint32_t blocks = (count + instances - 1) / instances;

  kernel_mul_reduce<<<blocks, threads>>>(report, res, in1, in2, module_data, tmp_buffer, inv, count);
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
  return 0;
}

}//gpu
